
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

typedef struct
{
   float3 position, velocity;
} Particle;

__host__ double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void distanceKernel(int NUM_PARTICLES, Particle *particles_GPU, float dt)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= NUM_PARTICLES) return;

        particles_GPU[i].velocity.x = 3.81* particles_GPU[i].velocity.x *(1 - particles_GPU[i].velocity.x);  //  updating velocities
        particles_GPU[i].velocity.y = 3.81* particles_GPU[i].velocity.y *(1 - particles_GPU[i].velocity.y);
        particles_GPU[i].velocity.z = 3.81* particles_GPU[i].velocity.z *(1 - particles_GPU[i].velocity.z);

        particles_GPU[i].position.x = particles_GPU[i].position.x + dt * particles_GPU[i].velocity.x;
        particles_GPU[i].position.y = particles_GPU[i].position.y + dt * particles_GPU[i].velocity.y;
        particles_GPU[i].position.z = particles_GPU[i].position.z + dt * particles_GPU[i].velocity.z;
}

int main(int argc, char* argv[])
{
    if (argc < 4) return;
    const int NUM_PARTICLES = strtol(argv[1], nullptr, 0);
    const int NUM_ITERATIONS = strtol(argv[2], nullptr, 0);
    const int BLOCK_SIZE = strtol(argv[3], nullptr, 0);
    const int SIMULATION_TYPE = strtol(argv[4], nullptr, 0);
    
    float dt = 1.;
    Particle *particles;
    if (SIMULATION_TYPE)
        particles = (Particle*)malloc(NUM_PARTICLES*sizeof(Particle));
    else
    {   
        hipHostMalloc((void **) &particles, NUM_PARTICLES*sizeof(Particle), hipHostMallocDefault, hipHostMallocDefault);
    }
    Particle *particles_GPU; 
    
    for (int i = 0; i < NUM_PARTICLES; i++)
    {
        particles[i].position.x = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        particles[i].position.y = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        particles[i].position.z = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        particles[i].velocity.x = abs(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
        particles[i].velocity.y = abs(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
        particles[i].velocity.z = abs(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
    }
    
    double iStart = cpuSecond();
    hipMalloc(&particles_GPU, NUM_PARTICLES*sizeof(Particle));
 
    for (int k = 0; k < NUM_ITERATIONS; k++)
    {    
        hipMemcpy(particles_GPU, particles, NUM_PARTICLES*sizeof(Particle), hipMemcpyHostToDevice);   
        distanceKernel<<<(NUM_PARTICLES + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE>>>(NUM_PARTICLES, particles_GPU, dt);
        hipDeviceSynchronize();
        hipMemcpy(particles, particles_GPU, NUM_PARTICLES*sizeof(Particle), hipMemcpyDeviceToHost);
    }
    hipFree(particles_GPU);

    double iElapsGPU = cpuSecond() - iStart;
    if (SIMULATION_TYPE)
        printf("Computing SAXPY on the GPU cudaMalloc… Done!\nIt took %f ms\n", iElapsGPU*1000);
    else
        printf("Computing SAXPY on the GPU cudaMallocHost… Done!\nIt took %f ms\n", iElapsGPU*1000);

    return 0;
}